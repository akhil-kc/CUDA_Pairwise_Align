#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "dna.h"
//#include "Alignment.cu"
#include <iostream>
#include "SWUtils.h"
//#include "KernelUtil.cu"
#include<hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
using namespace std;
//#include "KernelUtil.cu"
//#include "Alignment.cu"
#include <iostream>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 32
#define BLKYSIZE 4
#define BLKZSIZE 4

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)



__device__ int maxScore2(int score1,int score2) {
        return ((score1>score2)?score1:score2);
}

__device__ int maxScore3(int score1,int score2,int score3) {
        return ((score1>score2)?((score1>score3)?score1:score3):((score2>score3)?score2:score3));
}

__global__ void parallel_scan(DNA sequences[2],int *score,int *prev_row,unsigned iteration)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    int xLen=sequences[0].seq_length;
    int yLen=sequences[1].seq_length;
    unsigned index= idx + (xLen * idy );
    if (index < xLen){
        score[index]=(int)maxScore2(prev_row[index-1]+5,prev_row[index]-3);
      }
}

int main(int argc, char *argv[])
{
    DNA *sequences=new DNA[2];
    sequences[0].read_file(argv[1]);
    sequences[1].read_file(argv[2]);
    //Sequence 1 along the x-axis should always be greater
    if (sequences[0].seq_length<sequences[1].seq_length)
    {
	DNA temp;
	temp.seq_string=sequences[0].seq_string;
	temp.seq_length=sequences[0].seq_length;
	sequences[0].seq_string=sequences[1].seq_string;
	sequences[0].seq_length=sequences[1].seq_length;
	sequences[1].seq_string=temp.seq_string;
	sequences[1].seq_length=temp.seq_length;
     }
    printf("\n Sequence 1: %s \t Length: %d",sequences[0].seq_string,sequences[0].seq_length);
    printf("\n Sequence 2: %s \t Length: %d",sequences[1].seq_string,sequences[1].seq_length);
    const int nx = sequences[0].seq_length;
    const int ny = sequences[1].seq_length;
    unsigned size=nx*ny;
    int *c,*temp; 
    int *d_c;  
    int *final[ny];
cout<<"\n Before malloc";
    if ((c = (int *)malloc((nx*ny)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
cout<<"\n After malloc";
    for(int i=0;i<nx;i++)
	final[i][0]=(-i);
    for(int i=0;i<ny;i++)
	final[0][i]=(-i);
cout<<"\n After Init";
    
    
    hipMalloc((void **) &d_c, (nx)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    for (unsigned iteration=1;iteration<ny;iteration++)
    {
	memcpy(final[iteration-1],temp,nx);
    	parallel_scan<<<(nx/512),512>>>(sequences,d_c,temp,iteration);
    	cudaCheckErrors("Kernel launch failure");
    	hipMemcpy(c, d_c, ((nx)*sizeof(int)), hipMemcpyDeviceToHost);
    	cudaCheckErrors("CUDA memcpy failure");
	memcpy(final[iteration],c,nx);
    }
    free(c);
    hipFree(d_c);
    cudaCheckErrors("hipFree fail");
    cout<<"\n\nSUCCESS";
    return 0;
}
