#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "dna.h"
//#include "Alignment.cu"
#include <iostream>
#include "SWUtils.h"
//#include "KernelUtil.cu"
#include<hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
using namespace std;
//#include "KernelUtil.cu"
//#include "Alignment.cu"
#include <iostream>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 32
#define BLKYSIZE 4
#define BLKZSIZE 4

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)



__device__ int maxScore2(int score1,int score2) {
        return ((score1>score2)?score1:score2);
}

__device__ int maxScore3(int score1,int score2,int score3) {
        return ((score1>score2)?((score1>score3)?score1:score3):((score2>score3)?score2:score3));
}

__global__ void parallel_scan(char *sequence1,char *sequence2,int xLen,int yLen,int *score,int *prev_row,unsigned iteration)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned index= idx + (xLen * idy );
    if (index < xLen){
        score[index]=(int)maxScore2(prev_row[index-1]+5,prev_row[index]-3);
      }
}

int main(int argc, char *argv[])
{
    DNA sequence1,sequence2;
    sequence1.read_file(argv[1]);
    sequence2.read_file(argv[2]);
    char *d_sequence1,*d_sequence2;  
//Sequence 1 along the x-axis should always be greater
    if (sequence1.seq_length<sequence2.seq_length)
    {
	DNA temp;
	temp.seq_string=sequence1.seq_string;
	temp.seq_length=sequence1.seq_length;
	sequence1.seq_string=sequence2.seq_string;
	sequence1.seq_length=sequence2.seq_length;
	sequence2.seq_string=temp.seq_string;
	sequence2.seq_length=temp.seq_length;
     }
    printf("\n Sequence 1: %s \t Length: %d",sequence1.seq_string,sequence1.seq_length);
    printf("\n Sequence 2: %s \t Length: %d",sequence2.seq_string,sequence2.seq_length);
    const int nx = sequence1.seq_length;
    const int ny = sequence2.seq_length;
	    unsigned size=nx*ny;
    int *c,*temp; 
    int *d_c;  
    int **final;
    final = (int **) malloc ((nx)*sizeof(int));
    if ((c = (int *)malloc((nx*ny)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
    for(int i=0;i<nx;i++)
    {
	final[i] = (int *) malloc (ny * sizeof(int));
	final[i][0]=0;
    }
    for(int i=0;i<ny;i++)
	final[0][i]=0;
    hipMalloc((void **) &d_c, (nx)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_sequence1, (nx)*sizeof(char));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMemcpy(d_sequence1, &sequence1.seq_string, (nx)*sizeof(char), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy device buffer");

    hipMalloc((void **) &d_sequence2, (ny)*sizeof(char));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMemcpy(d_sequence2, &sequence2.seq_string, (ny)*sizeof(char), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy device buffer");
    for (unsigned iteration=1;iteration<ny;iteration++)
    {
	cout<<"\n Iteration"<<iteration<<endl;
	memcpy(temp,final[iteration-1],nx);
    	parallel_scan<<<(nx/512),512>>>(d_sequence1,d_sequence2,nx,ny,d_c,temp,iteration);
    	cudaCheckErrors("Kernel launch failure");
	hipDeviceSynchronize();
    	hipMemcpy(final[iteration], d_c, ((nx)*sizeof(int)), hipMemcpyDeviceToHost);
    	cudaCheckErrors("CUDA memcpy failure");
//	memcpy(final[iteration],c,nx);
    }
    free(c);
    hipFree(d_c);
    cudaCheckErrors("hipFree fail");
    cout<<"\n\nSUCCESS";
    return 0;
}
